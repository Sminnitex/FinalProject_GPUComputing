#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <math.h>
#include "./include/library.h"
#include "./include/hip/hip_runtime_api.h"
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#define NDEVICE 4
#define TIMER_DEF     struct timeval temp_1, temp_2
#define TIMER_START   gettimeofday(&temp_1, (struct timezone*)0)
#define TIMER_STOP    gettimeofday(&temp_2, (struct timezone*)0)
#define TIMER_ELAPSED ((temp_2.tv_sec-temp_1.tv_sec)+(temp_2.tv_usec-temp_1.tv_usec)/1000000.0)

int main(int argc, char *argv[]) {
    //The paths of our benchmark matrices
    const char* path[] = {
        "./dataset/1138_bus/1138_bus.mtx",
        "./dataset/Maragal_3/Maragal_3.mtx",
        "./dataset/photogrammetry/photogrammetry.mtx",
        "./dataset/plbuckle/plbuckle.mtx",
        "./dataset/bcsstk17/bcsstk17.mtx",
        "./dataset/filter2D/filter2D.mtx",
        "./dataset/SiH4/SiH4.mtx",
        "./dataset/linverse/linverse.mtx",
        "./dataset/t2dah_a/t2dah_a.mtx",
        "./dataset/barrier2-10/barrier2-10.mtx"
    };

    //Stats of my problem
    srand(time(NULL));
    int array_length = sizeof(path) / sizeof(path[0]);  
    int blocksize = 16;
    int gridsize = 7;
    printf("==============================================================\n");
    printf("STATS OF MY PROBLEM\n");
    printf("block size = %d \n", blocksize);
    printf("grid size = %d \n", gridsize);
    dim3 block_size(blocksize, blocksize, 1);
    dim3 grid_size(gridsize, gridsize, 1);
    printf("%d: block_size = (%d, %d), grid_size = (%d, %d)\n", __LINE__, block_size.x, block_size.y, grid_size.x, grid_size.y);
    int sharedMemSize = sizeof(dtype) * block_size.x * block_size.y * 2;
    int *number, m, n, nnz;
    int nnz_counter;

    //Timer definitions
    TIMER_DEF;
    float times[NDEVICE] = {0};
    char filename[556];
    FILE *csvtime[array_length];

    //Print device properties for unitn cluster
    FILE *file = fopen("warp.txt", "r");
    if (file == NULL) {
        printf("Error opening file!\n");
        return 1;
    }
        
    printf("==============================================================\n");
    printf("DEVICE PROPERTIES\n");
    char ch;
    while ((ch = fgetc(file)) != EOF) {
        printf("%c", ch);
    }
    fclose(file);

    for(int k = 0; k < array_length; k++){
        //Initialize all the stuff we need
        dtype *matrix = NULL;
        number = (int *)malloc(3 * sizeof(int));
        read_mtx(path[k], matrix, number);
        nnz_counter = 0;
        
        //Assign number of rows, columns and non zero elements
        m = number[0]; 
        n = number[1]; 
        nnz = number[2];  

        //Initialize kernel
        dummyKernel<<<grid_size, block_size>>>();
        checkCudaErrors(hipGetLastError()); 
        checkCudaErrors(hipDeviceSynchronize());

        //Cusparse handle and stream
        hipsparseHandle_t handle;
        hipsparseCreate(&handle);
        checkCudaErrors(hipGetLastError()); 
        hipStream_t stream;
        checkCudaErrors(hipStreamCreate(&stream));


        //Allocate and initialize host memory
        int *h_csrRowPtr = (int *)malloc((m + 1) * sizeof(int));
        int *h_csrColInd = (int *)malloc(nnz * sizeof(int));
        dtype *h_csrVal = (dtype *)malloc(nnz * sizeof(dtype));

        if (h_csrRowPtr == NULL || h_csrColInd == NULL || h_csrVal == NULL) {
           fprintf(stderr, "Error allocating host memory\n");
           return 1;
        }

        //Allocate device memory
        int *d_csrRowPtr, *d_csrColInd, *d_cscRowInd, *d_cscColPtr;
        dtype *d_csrVal, *d_cscVal;
        checkCudaErrors(hipMalloc((void **)&d_csrRowPtr, (m + 1) * sizeof(int)));
        checkCudaErrors(hipMalloc((void **)&d_csrColInd, nnz * sizeof(int)));
        checkCudaErrors(hipMalloc((void **)&d_csrVal, nnz * sizeof(dtype)));
        checkCudaErrors(hipMalloc((void **)&d_cscRowInd, nnz * sizeof(int)));
        checkCudaErrors(hipMalloc((void **)&d_cscColPtr, (n + 1) * sizeof(int)));
        checkCudaErrors(hipMalloc((void **)&d_cscVal, nnz * sizeof(dtype)));

        //Assign values host memory
        h_csrRowPtr[0] = 0;
        for (int i = 0; i < m; i++) {
            for (int j = 0; j < n; j++) {
                if (matrix[i + j * m] != 0) {
                    if (nnz_counter >= nnz) {
                        fprintf(stderr, "Error: nnz_counter (%d) exceeded nnz (%d) at i=%d, j=%d\n", nnz_counter, nnz, i, j);
                        exit(EXIT_FAILURE);
                    }
                    h_csrColInd[nnz_counter] = j;
                    h_csrVal[nnz_counter] = matrix[i + j * m]; 
                    nnz_counter++;
                }
            }
        if (i + 1 > m) {
            fprintf(stderr, "Error: i+1 (%d) exceeded m (%d)\n", i + 1, m);
            exit(EXIT_FAILURE);
        }
        h_csrRowPtr[i + 1] = nnz_counter;
    }
        
        //Copy data to device
        checkCudaErrors(hipMemcpyAsync(d_csrRowPtr, h_csrRowPtr, (m + 1) * sizeof(int), hipMemcpyHostToDevice, stream));
        checkCudaErrors(hipMemcpyAsync(d_csrColInd, h_csrColInd, nnz * sizeof(int), hipMemcpyHostToDevice, stream));
        checkCudaErrors(hipMemcpyAsync(d_csrVal, h_csrVal, nnz * sizeof(dtype), hipMemcpyHostToDevice, stream));
        
        //Perform sparse matrix transpose with cusparse
        void *buffer;
        cusparseBuffer(handle, m, n, nnz, d_csrVal, d_csrRowPtr, d_csrColInd, 
                            d_cscVal, d_cscColPtr, d_cscRowInd, buffer);
        TIMER_START;
        cusparseTranspose(handle, m, n, nnz, d_csrVal, d_csrRowPtr, d_csrColInd, 
                            d_cscVal, d_cscColPtr, d_cscRowInd, buffer);
        TIMER_STOP;
        times[0] = TIMER_ELAPSED;
        
        //Copy the transposed matrix back to host
        checkCudaErrors(hipMemcpy(h_csrRowPtr, d_cscColPtr, (n + 1) * sizeof(int), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(h_csrColInd, d_cscRowInd, nnz * sizeof(int), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(h_csrVal, d_cscVal, nnz * sizeof(dtype), hipMemcpyDeviceToHost));

        //Perform a normal matrix transpose with kernels from homework 2
        dtype *transpose = NULL, *transposeShared = NULL, *d_matrix = NULL;
        checkCudaErrors(hipMallocManaged(&d_matrix, sizeof(dtype) * m * n ));
        checkCudaErrors(hipMallocManaged(&transpose, sizeof(dtype) * m * n));
        checkCudaErrors(hipMallocManaged(&transposeShared, sizeof(dtype) * m * n));
        checkCudaErrors(hipMemcpy(d_matrix, matrix, sizeof(dtype) * m * n , hipMemcpyHostToDevice));

        //Global matrix transpose
        TIMER_START;
        transposeGlobalMatrix<<<grid_size, block_size, sharedMemSize, stream>>>(d_matrix, transpose, m, n);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());
        checkCudaErrors(hipStreamSynchronize(stream));
        TIMER_STOP;
        times[1] = TIMER_ELAPSED;

        //Shared matrix transpose
        TIMER_START;
        transposeSharedMatrix<<<grid_size, block_size, sharedMemSize, stream>>>(d_matrix, transposeShared, m, n);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());
        checkCudaErrors(hipStreamSynchronize(stream));
        TIMER_STOP;
        times[2] = TIMER_ELAPSED;

        //Copy back to host for debug purposes
        dtype *h_transpose = NULL, *h_transposeShared = NULL;
        h_transpose = (dtype*)malloc(m * n * sizeof(dtype));
        h_transposeShared = (dtype*)malloc(m * n * sizeof(dtype));
        checkCudaErrors(hipMemcpy(h_transpose, transpose, m * n * sizeof(dtype), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(h_transposeShared, transposeShared, m * n * sizeof(dtype), hipMemcpyDeviceToHost));

        //Perform a transpose with kernel adapted for sparse matrices
        int *d_my_csrRowPtr, *d_my_csrColInd, *d_my_cscRowInd, *d_my_cscColPtr;
        dtype *d_my_csrVal, *d_my_cscVal;
        checkCudaErrors(hipMalloc((void **)&d_my_csrRowPtr, (m + 1) * sizeof(int)));
        checkCudaErrors(hipMalloc((void **)&d_my_csrColInd, nnz * sizeof(int)));
        checkCudaErrors(hipMalloc((void **)&d_my_csrVal, nnz * sizeof(dtype)));
        checkCudaErrors(hipMalloc((void **)&d_my_cscRowInd, nnz * sizeof(int)));
        checkCudaErrors(hipMalloc((void **)&d_my_cscColPtr, (n + 1) * sizeof(int)));
        checkCudaErrors(hipMalloc((void **)&d_my_cscVal, nnz * sizeof(dtype)));

        checkCudaErrors(hipMemcpy(d_my_csrRowPtr, h_csrRowPtr, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(d_my_csrColInd, h_csrColInd, nnz * sizeof(int), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(d_my_csrVal, h_csrVal, nnz * sizeof(dtype), hipMemcpyHostToDevice));
         
        TIMER_START;
        sparseMatrixTranspose(m, n, nnz, d_my_csrVal, d_my_csrRowPtr, d_csrColInd, d_my_cscVal, d_my_cscColPtr, d_my_cscRowInd, sharedMemSize, stream);
        TIMER_STOP;
        times[3] = TIMER_ELAPSED;

        //Print effective Bandwidth
        printf("==============================================================\n");
        printf("STATS of %s\n", path[k]);
        printf("Sparse Matrix Transpose With Cusparse Effective Bandwidth(GB/s): %f\n", (2 * m * n * sizeof(dtype)) / (1e9 * times[0]));
        printf("Global Matrix Transpose Effective Bandwidth(GB/s): %f\n", (2 * m * n * sizeof(dtype)) / (1e9 * times[1]));
        printf("Shared Matrix Transpose Effective Bandwidth(GB/s): %f\n", (2 * m * n * sizeof(dtype)) / (1e9 * times[2]));
        printf("My Sparse Matrix Transpose Effective Bandwidth(GB/s): %f\n", (2 * m * n * sizeof(dtype)) / (1e9 * times[3]));

        //Produce output files
        sprintf(filename, "output/Matrix%d.csv", k);
        csvtime[k] = fopen(filename, "w");
        if (csvtime[k] == NULL) {
            printf("Error opening file!\n");
            return 1;
        }
        printf("%c", filename[k]);
        fprintf(csvtime[k], "TheRowsAre,Legend,Cusparse,Global,Shared,MySparse\n");
        fprintf(csvtime[k], "Bandwidth,Rows,Columns,NonZeros\n");

        for (int i = 0; i < 4; i++) {
            fprintf(csvtime[k], "%f,%d,%d,%d\n", (2 * m * n * sizeof(dtype)) / (1e9 * times[i]), m, n, nnz);
        }

        fclose(csvtime[k]);

        //Lines for debug purposes
        //printMatrix(matrix, m, n, "Matrix");
        //printSparseMatrix(h_csrRowPtr, h_csrColInd, h_csrVal, n, nnz, "Cusparse Transposed Matrix");
        //printMatrix(h_transpose, m, n, "Transpose");
        //printMatrix(h_transposeShared, m, n, "Transpose Shared");
        //printDeviceData<<<1, 1, 0, stream>>>(d_cscColPtr, d_cscRowInd, d_csrVal, m, nnz);

        //Destroy everything
        checkCudaErrors(hipFree(d_csrRowPtr));
        checkCudaErrors(hipFree(d_csrColInd));
        checkCudaErrors(hipFree(d_csrVal));
        checkCudaErrors(hipFree(d_cscRowInd));
        checkCudaErrors(hipFree(d_cscColPtr));
        checkCudaErrors(hipFree(d_cscVal));
        checkCudaErrors(hipFree(d_matrix));
        hipsparseDestroy(handle);
        checkCudaErrors(hipFree(transpose));
        checkCudaErrors(hipFree(transposeShared));
        checkCudaErrors(hipStreamDestroy(stream));

        checkCudaErrors(hipFree(d_my_csrRowPtr));
        checkCudaErrors(hipFree(d_my_csrColInd));
        checkCudaErrors(hipFree(d_my_csrVal));
        checkCudaErrors(hipFree(d_my_cscRowInd));
        checkCudaErrors(hipFree(d_my_cscColPtr));
        checkCudaErrors(hipFree(d_my_cscVal));

        free(h_csrRowPtr);
        free(h_csrColInd);
        free(h_csrVal);
        free(matrix);
        free(h_transpose);
        free(h_transposeShared);
        free(number);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceReset());
        
        //Reset times array
        for (int i = 0; i < NDEVICE; i++) {
            times[i] = 0.0;
        }
    }

    return 0;
}
